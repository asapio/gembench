#include "hip/hip_runtime.h"
/*******************************************************************************
@ddblock_begin copyright

Copyright (c) 1997-2019
Maryland DSPCAD Research Group, The University of Maryland at College Park 

Permission is hereby granted, without written agreement and without license or
royalty fees, to use, copy, modify, and distribute this software and its
documentation for any purpose other than its incorporation into a commercial
product, provided that the above copyright notice and the following two
paragraphs appear in all copies of this software.

IN NO EVENT SHALL THE UNIVERSITY OF MARYLAND BE LIABLE TO ANY PARTY
FOR DIRECT, INDIRECT, SPECIAL, INCIDENTAL, OR CONSEQUENTIAL DAMAGES
ARISING OUT OF THE USE OF THIS SOFTWARE AND ITS DOCUMENTATION, EVEN IF
THE UNIVERSITY OF MARYLAND HAS BEEN ADVISED OF THE POSSIBILITY OF
SUCH DAMAGE.

THE UNIVERSITY OF MARYLAND SPECIFICALLY DISCLAIMS ANY WARRANTIES,
INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF
MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE. THE SOFTWARE
PROVIDED HEREUNDER IS ON AN "AS IS" BASIS, AND THE UNIVERSITY OF
MARYLAND HAS NO OBLIGATION TO PROVIDE MAINTENANCE, SUPPORT, UPDATES,
ENHANCEMENTS, OR MODIFICATIONS.

@ddblock_end copyright
*******************************************************************************/

// C
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// CUDA Runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "hipsparse.h"

// CUDA files
#include "cuda_init.h"

// Solver// File parser interfaces
#include "pomdpCassandraWrapper.h"

// Solver interfaces
#include "solver_spvi.h"

// Misc files
#include "utils.h"

// #define ALLOW_PRINTS

// TEMP - Load these into ram for now

static int    s_nnz = 0;
static size_t s_Ns = 0;
static size_t s_Na = 0;
static size_t s_NsNa = 0;     // Shorthand for "Ns times Na"
static size_t s_Ns2Na = 0;    // Shorthand for "Ns squared times Na"
static float s_discount_factor = 0;
static float s_stopping_thresh = 0;

// Pointers to buffers in CPU RAM
static int*   s_host_cooRowIndex = NULL;
static int*   s_host_cooColIndex = NULL;
static float* s_host_cooVal = NULL;

// Pointers to buffers in GPU
static float* s_dev_PV;
static float* s_dev_CV;
static int*   s_dev_CP;
static float* s_dev_Q;
static float* s_dev_R;
static int*   s_dev_cooRowIndex;
static int*   s_dev_cooColIndex;
static float* s_dev_cooVal;
static int*   s_dev_csrRowPtr=0;

static hipsparseHandle_t s_handle = 0;
static hipsparseMatDescr_t s_stms_descr=0;


// Memory using in sup_norm reduction kernel
// Needs file scope so we can free the malloc'd memory
// after the solver completes
static float* s_h_reduce_out_vec = NULL;
static float* s_d_reduce_out_vec = NULL;

__global__
void select_best_action(int num_states, int num_actions, const float *dev_Q, float *dev_CV, int* dev_CP)
{
    int n = blockIdx.x*blockDim.x + threadIdx.x;

    // More kernels than states will be launched, dont go out of bounds
    if (n < num_states)
    {
        float max_value = -1e6;
        int32_t best_action = -1;

        for (int a_idx=0; a_idx<num_actions; a_idx++)
        {
            // Compute index in Q
            int32_t q_index = a_idx*(num_states) + n;

            float value_for_this_action = dev_Q[q_index];

            // Is this the new best action?
            if (value_for_this_action > max_value)
            {
                max_value = value_for_this_action;
                best_action = a_idx;
            }
        }

        dev_CV[n] = max_value;
        dev_CP[n] = best_action;
    }
}

// Reduction kernel taken from "reduction" example in CUDA samples.
// More info can be found here:
// http://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf
// This is the "#4" example in the presentation

__global__
void reduce_sup_norm(const float *g_idata_1, const float *g_idata_2, float *g_odata, unsigned int n)
{
    extern __shared__ float sdata[];

//    // perform first level of reduction,
//    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

//    T mySum = (i < n) ? g_idata[i] : 0;
    float myMaxDelta = (i < n) ? fabsf(g_idata_1[i]-g_idata_2[i]) : 0.0f;

    if ((i + blockDim.x) < n)
    {
//        mySum += g_idata[i+blockDim.x];
        float newDelta = fabsf(g_idata_1[i+blockDim.x]-g_idata_2[i+blockDim.x]);
        myMaxDelta = newDelta > myMaxDelta ? newDelta : myMaxDelta;
    }

    sdata[tid] = myMaxDelta;
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
        if (tid < s)
        {
//            sdata[tid] = mySum = mySum + sdata[tid + s];
            float newDelta = sdata[tid + s];
            myMaxDelta = newDelta > myMaxDelta ? newDelta : myMaxDelta;
            sdata[tid] = myMaxDelta;
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0)
    {
//        g_odata[blockIdx.x] = mySum;
        g_odata[blockIdx.x] = myMaxDelta;
    }
}



static void solver_do_backup(
        const float* dev_R,
        const float* dev_PV,
        float* dev_CV,
        int* dev_CP,
        float* dev_Q)
{
    static const float fOne = 1.0f;

    hipError_t cudaErr;

    // Copy dev_R into dev_Q
    cudaErr = hipMemcpy(dev_Q, dev_R, (size_t)(s_NsNa*sizeof(float)), hipMemcpyDeviceToDevice);
    assert(cudaErr == hipSuccess);

    float alpha = s_discount_factor;

    // Multiply Matrix times vector
    hipsparseStatus_t status;
    status = hipsparseScsrmv(s_handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            s_NsNa,                    // int m, Rows in Matrix
            s_Ns,                       // int n, Cols in Matrix
            s_nnz,                      // int nnz, # of Non-Zero elements in Matrix
            &alpha,                     // const float *alpha, // Addition constant
            s_stms_descr,               // const hipsparseMatDescr_t descrA, // Matrix descriptor
            s_dev_cooVal,                 // const float *csrValA, // Values
            s_dev_csrRowPtr,              // const int *csrRowPtrA, // CSR format row pointer
            s_dev_cooColIndex,            // const int *csrColIndA, // CSR format col indicies
            &dev_PV[0],                 // const float *x,
            &fOne,                      // const float *beta,   // Addition constant
            &dev_Q[0]);                 // float *y);   //
    assert(status == HIPSPARSE_STATUS_SUCCESS);

    // Select best action using CUDA kernel
    // Launch 1 kernel per MDP state
    // Use thread blocks with 256 threads per thread block
    select_best_action<<<(s_Ns+255)/256, 256>>>(s_Ns, s_Na, dev_Q, dev_CV,dev_CP);

    hipDeviceSynchronize();
}

float compute_sup_norm(const float* dev_v1,
                       const float* dev_v2,
                       uint32_t N)
{
//    static int kernel_num_blocks = (N+255)/256;
    static int kernel_num_blocks = (N+255)/(256*2); // Need half the blocks due to optimization in kernel
    static int kernel_num_threads = 256;

    hipError_t cudaErr;

    // #warning "Temp hack using CPU Sup Norm!"

    // USE CPU version for now
    if (kernel_num_blocks == 0)
    {
        // N here is <= 256
        float host_v1[N];
        float host_v2[N];

        cudaErr = hipMemcpy(host_v1, dev_v1, (size_t)(N*sizeof(float)), hipMemcpyDeviceToHost);
        assert(cudaErr == hipSuccess);
        cudaErr = hipMemcpy(host_v2, dev_v2, (size_t)(N*sizeof(float)), hipMemcpyDeviceToHost);
        assert(cudaErr == hipSuccess);

        float max_abs_delta = 0.0f;
        float abs_delta;
        for (uint32_t n=0; n<N; n++)
        {
            abs_delta = fabsf(host_v1[n]-host_v2[n]);
            if (abs_delta > max_abs_delta)
            {
                // printf("[%d] %f > %f, new_max_abs_delta\n", n, abs_delta, max_abs_delta);
                max_abs_delta = abs_delta;
            }
        }
        return max_abs_delta;
    }
    else
    {
        // USE GPU VERSION
        if (s_h_reduce_out_vec == NULL)
        {
            // TODO - We could allocate these earlier. Doing them here since size is a function of
            // kernel_num_blocks

            #ifdef ALLOW_PRINTS
            printf("N = %d, NB = %d, NT = %d\n", N, kernel_num_blocks, kernel_num_threads);
            #endif

            s_h_reduce_out_vec = (float*)malloc(sizeof(float)*kernel_num_blocks);
            assert(s_h_reduce_out_vec != NULL);
        }
        if (s_d_reduce_out_vec == NULL)
        {

            cudaErr = hipMalloc((void**)&s_d_reduce_out_vec, kernel_num_blocks*sizeof(float));
            assert(cudaErr == hipSuccess);
        }

        // Do first stage reduction using CUDA kernel
        // This leaves a length kernel_num_blocks array that needs to still be reduced
        reduce_sup_norm<<<kernel_num_blocks, kernel_num_threads, kernel_num_threads*sizeof(float)>>>(dev_v1, dev_v2, s_d_reduce_out_vec, N);
        hipDeviceSynchronize();

        cudaErr = hipMemcpy(s_h_reduce_out_vec, s_d_reduce_out_vec, (size_t)(kernel_num_blocks*sizeof(float)), hipMemcpyDeviceToHost);
        checkCudaErrors(cudaErr);
        assert(cudaErr == hipSuccess);

        float temp_max = 0.0f;
        for (int n=0; n<kernel_num_blocks; n++)
        {
            if (s_h_reduce_out_vec[n] > temp_max)
            {
                temp_max = s_h_reduce_out_vec[n];
            }
        }

        //printf("CPU,CUDA sup_norm = %f %f\t", max_abs_delta, temp_max);
        return temp_max;
    }

}

// This function currently assumes that the input format is the cassandra format
// It converts the cassandra format to the MDP format that this solver uses
// The converted mdp variables have file scope.
// The intention is to handle other incoming formats here as well
static void change_mdp_format(void* p_mdp_obj)
{
    PomdpCassandraWrapper* p_mdp = (PomdpCassandraWrapper*)p_mdp_obj;
    s_discount_factor = p_mdp->getDiscount();
    s_Ns = p_mdp->getNumStates();
    s_Na = p_mdp->getNumActions();

    s_NsNa = s_Ns*s_Na;
    s_Ns2Na = s_Ns*s_Ns*s_Na;

    float eps = 0.5f;
    s_stopping_thresh = (eps * (1-s_discount_factor)) / (2*s_discount_factor);

    // -------------------------------------
    // Load MDP STM,R into Host RAM
    // -------------------------------------
    // Populate STMs in COO format

    s_nnz = 0;
    for(uint32_t a_idx=0; a_idx<s_Na; a_idx++)
    {
        CassandraMatrix single_stm = p_mdp->getT(a_idx);
        s_nnz += single_stm->num_non_zero;
    }

    printf("Total non-zero entries = %d / %lu (= %.3f %% Sparse)\n",
           s_nnz, s_Ns2Na, 100.0f*((float)(s_Ns2Na-s_nnz))/(float(s_Ns2Na)));

    s_host_cooRowIndex = (int*)malloc(s_nnz*sizeof(int));
    s_host_cooColIndex = (int*)malloc(s_nnz*sizeof(int));
    s_host_cooVal =    (float*)malloc(s_nnz*sizeof(float));

    uint32_t count = 0;
    for(uint32_t a_idx=0; a_idx<s_Na; a_idx++)
    {
        CassandraMatrix single_stm = p_mdp->getT(a_idx);
        // displayMatrix(single_stm);
        for (uint32_t s_idx=0; s_idx<s_Ns; s_idx++)
        {
            for (uint32_t next_s_idx=0; next_s_idx<s_Ns; next_s_idx++)
            {
                float transition_prob = getEntryMatrix(single_stm, s_idx, next_s_idx);
                if (transition_prob > 0.0f)
                {
                    assert(count < s_nnz);
                    s_host_cooRowIndex[count] = s_idx + a_idx*s_Ns;
                    s_host_cooColIndex[count] = next_s_idx;
                    s_host_cooVal[count] = transition_prob;
                    count++;
                }
            }
        }
    }
    assert(count == s_nnz);

    // Populate R in full matrix format
    float* R_2D_lut = (float*)malloc(sizeof(float)*s_NsNa);
    memset(R_2D_lut, 0, sizeof(float)*s_NsNa);

    uint32_t r_idx = 0;
    CassandraMatrix cassandra_RTranspose = p_mdp->getRTranspose();
    // displayMatrix(cassandra_RTranspose);
    for(uint32_t a_idx=0; a_idx<s_Na; a_idx++)
    {
        for(uint32_t s_idx=0; s_idx<s_Ns; s_idx++)
        {
            float reward = getEntryMatrix(cassandra_RTranspose, a_idx, s_idx);
            R_2D_lut[r_idx] = reward;
            // printf("[%d] : R(%d, %d) <= %f\n", r_idx, a_idx, s_idx, reward);
            r_idx++;
        }
    }

    // -------------------------------------
    // Allocate Storage on device
    // -------------------------------------
    hipError_t cudaStat;

    // Previous value function (init to zero)
    cudaStat = hipMalloc((void**)&s_dev_PV, s_Ns*sizeof(float));
    assert(cudaStat == hipSuccess);

    cudaStat = hipMemset(s_dev_PV, 0, s_Ns);
    assert(cudaStat == hipSuccess);

    cudaStat = hipMalloc((void**)&s_dev_CV, s_Ns*sizeof(float));
    assert(cudaStat == hipSuccess);

    cudaStat = hipMalloc((void**)&s_dev_CP, s_Ns*sizeof(int));
    assert(cudaStat == hipSuccess);

    cudaStat = hipMalloc((void**)&s_dev_Q, s_NsNa*sizeof(float));
    assert(cudaStat == hipSuccess);

    // Rewards
    cudaStat = hipMalloc((void**)&s_dev_R, s_NsNa*sizeof(float));
    assert(cudaStat == hipSuccess);

    // STMs
    cudaStat = hipMalloc((void**)&s_dev_cooRowIndex, s_nnz*sizeof(int));
    assert(cudaStat == hipSuccess);

    cudaStat = hipMalloc((void**)&s_dev_cooColIndex, s_nnz*sizeof(int));
    assert(cudaStat == hipSuccess);

    cudaStat = hipMalloc((void**)&s_dev_cooVal, s_nnz*sizeof(float));
    assert(cudaStat == hipSuccess);

    cudaStat = hipMalloc((void**)&s_dev_csrRowPtr,(s_Ns+1)*sizeof(int));
    assert(cudaStat == hipSuccess);

    // -------------------------------------
    // Copy data to device
    // -------------------------------------

    // Copy STM from host to device
    cudaStat = hipMemcpy(s_dev_cooRowIndex, s_host_cooRowIndex, (size_t)(count*sizeof(int)), hipMemcpyHostToDevice);
    assert(cudaStat == hipSuccess);

    cudaStat = hipMemcpy(s_dev_cooColIndex, s_host_cooColIndex, (size_t)(count*sizeof(int)), hipMemcpyHostToDevice);
    assert(cudaStat == hipSuccess);

    cudaStat = hipMemcpy(s_dev_cooVal, s_host_cooVal, (size_t)(count*sizeof(float)), hipMemcpyHostToDevice);
    assert(cudaStat == hipSuccess);

    // Copy rewards from host to device
    const float* host_R = R_2D_lut;
    cudaStat = hipMemcpy(s_dev_R, host_R, (size_t)(s_NsNa*sizeof(float)), hipMemcpyHostToDevice);
    assert(cudaStat == hipSuccess);

    // Dont need this anymore. Free it.
    if (R_2D_lut != NULL) {free(s_h_reduce_out_vec);}

    // -------------------------------------
    // Init cuSpare library and structures
    // -------------------------------------
    hipsparseStatus_t status = hipsparseCreate(&s_handle);
    if (status != HIPSPARSE_STATUS_SUCCESS)
    {
        printf("CUSPARSE Library initialization failed");
        assert(false);
    }

    // create and setup matrix descriptor
    status = hipsparseCreateMatDescr(&s_stms_descr);
    if (status != HIPSPARSE_STATUS_SUCCESS)
    {
        printf("Matrix descriptor initialization failed");
        assert(false);
    }
    hipsparseSetMatType(s_stms_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(s_stms_descr,HIPSPARSE_INDEX_BASE_ZERO);

    // Transform STMs from COO to CSR format
    status = hipsparseXcoo2csr(s_handle,
            s_dev_cooRowIndex,
            count,
            s_Ns,
            s_dev_csrRowPtr,
            HIPSPARSE_INDEX_BASE_ZERO);
    assert(status == HIPSPARSE_STATUS_SUCCESS);
}

int solver_spvi_solve(void* p_mdp_obj, uint32_t* p_out_policy, float* p_out_value_func, int max_solver_time_s)
{
    printf("Solver spvi\n");

    assert(cuda_init(0) == EXIT_SUCCESS);

    // Load in MDP from external format
    change_mdp_format(p_mdp_obj);


    // printf("Starting Value Iteration\n");

    struct timespec start_time, elapsed_time;
    clock_gettime(CLOCK_MONOTONIC_RAW, &start_time);

    bool b_done = false;
    uint32_t num_iterations = 0;
    bool b_timed_out = false;
    while(!b_done)
    {
        num_iterations++;
        solver_do_backup(
                s_dev_R,
                s_dev_PV,
                s_dev_CV,
                s_dev_CP,
                s_dev_Q);

        // Compute stopping criteria
        float sup_norm = compute_sup_norm((const float*)s_dev_CV, (const float*)s_dev_PV, (uint32_t)s_Ns);

        if (sup_norm < s_stopping_thresh)
        {
            // Done
            b_done = true;
            printf("Iteration %d: %f < %f (STOP)\n", num_iterations, sup_norm, s_stopping_thresh);
        }
        else
        {
            // Check for time out
            if (max_solver_time_s != 0)
            {
                clock_gettime(CLOCK_MONOTONIC_RAW, &elapsed_time);

                float solver_elapsed_time = measure_elapsed_time(
                        (const struct timespec*)&start_time, (const struct timespec*)&elapsed_time);

//                printf("(%.1f of %d[s]) : ", solver_elapsed_time, max_solver_time_s);

                if ((int)solver_elapsed_time >= max_solver_time_s)
                {
                    b_done = true;
                    b_timed_out = true;
                }
            }
            //            printf("Iteration %d : %f > %f\n", num_iterations, sup_norm, s_stopping_thresh);
        }

        //        if (num_iterations == 2) b_done = true;

        // The value function computed in this iteration now becomes the "previous" value function.
        hipError_t cudaErr;
        cudaErr = hipMemcpy(s_dev_PV, s_dev_CV, (size_t)(s_Ns*sizeof(float)), hipMemcpyDeviceToDevice);
        assert(cudaErr == hipSuccess);
    }

    // Done. Save off policy and value
    hipError_t cudaErr;
    cudaErr = hipMemcpy(p_out_policy, s_dev_CP, (size_t)(s_Ns*sizeof(int)), hipMemcpyDeviceToHost);
    assert(cudaErr == hipSuccess);

    cudaErr = hipMemcpy(p_out_value_func, s_dev_CV, (size_t)(s_Ns*sizeof(float)), hipMemcpyDeviceToHost);
    assert(cudaErr == hipSuccess);


    // Free any CPU RAM that was malloc'd in this function
    if (s_host_cooRowIndex != NULL) {free(s_host_cooRowIndex);}
    if (s_host_cooColIndex != NULL) {free(s_host_cooColIndex);}
    if (s_host_cooVal != NULL) {free(s_host_cooVal);}
    if (s_h_reduce_out_vec != NULL) {free(s_h_reduce_out_vec);}


    // Free all GPU memory allocations
    assert(cuda_deinit() == EXIT_SUCCESS);

    if (b_timed_out)
    {
        return(1);
    }
    else
    {
        return(0);
    }

}


